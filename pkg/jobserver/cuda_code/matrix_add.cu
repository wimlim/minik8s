#include <iostream>
#include <hip/hip_runtime.h>


using namespace std;
const int M = 8;
const int N = 8;

__global__ void matrix_add(int *A, int *B, int *C, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int index = row * width + col;
    if (row < M && col < N) {
        C[index] = A[index] + B[index];
    }
}

int main() {
    const int num_elements = M * N;
    const int nbytes = num_elements * sizeof(int);
    
    int *host_A = (int *)malloc(nbytes);
    int *host_B = (int *)malloc(nbytes);
    int *host_C = (int *)malloc(nbytes);

    for (int i = 0; i < num_elements; i++) {
        host_A[i] = i;
        host_B[i] = i;
    }

    int *dev_A, *dev_B, *dev_C;
    hipMalloc((void **)&dev_A, nbytes);
    hipMalloc((void **)&dev_B, nbytes);
    hipMalloc((void **)&dev_C, nbytes);

    hipMemcpy(dev_A, host_A, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(dev_B, host_B, nbytes, hipMemcpyHostToDevice);
    hipMemset(dev_C, 0, nbytes);

    dim3 threadsPerBlock(8, 8);
    dim3 numBlocks(N / threadsPerBlock.x, M / threadsPerBlock.y);
    
    matrix_add<<<numBlocks, threadsPerBlock>>>(dev_A, dev_B, dev_C, N);

    hipDeviceSynchronize();
    hipMemcpy(host_C, dev_C, nbytes, hipMemcpyDeviceToHost);
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            cout << host_C[i * N + j] << " ";
        }
        cout << endl;
    }

    free(host_A);
    free(host_B);
    free(host_C);
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);
    return 0;
}
